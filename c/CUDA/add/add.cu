/*-==========================================================================-*\

Program Description
===================

This program demonstrates basic usage of the CUDA toolkit. `a` and `b` are
defined on the host, and these two values are passed to the device. The device
sums the values, and the result is copied back to the host.

Notes
=====

The hierarchy for structuring CUDA threads is: grid -> block -> thread. A kernel
has three predefined data structures available to it: `blockIDx`, `blockDim`,
and `threadIdx`.

`blockIDx` contains the index of the current block within it's grid. A grid is
two-dimensional, so available struct members are:

* blockIDx.x
* blockIDx.y

`blockDim` contains number of threads in the current block. A block is
three-dimensional, so available struct members are:

* blockDim.x
* blockDim.y
* blockDim.z

`threadIdx` contains the index of the current thread within it's block. Struct
members are:

* threadIDx.x
* threadIDx.y
* threadIDx.z

\*-==========================================================================-*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// c = a + b
__global__ void add(
    const int a, // input
    const int b, // input
    int * const c // output
) {
    *c = a + b;
}

// Calls `add` and prints the results of that call.
int main(void) {
    // Define and allocate host variables.
    const int a = 2;
    const int b = 7;
    int c = 1234; // The printf() statement at the end should not output 1234.

    // Define and allocate device variables.
    int * dev_c;
    hipMalloc(&dev_c, sizeof(int));

    // Perform business logic.
    // blocks per grid, threads per block
    add<<<1, 1>>>(a, b, dev_c);
    // dest, src, amount, operation
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d = %d\n", a, b, c);

    // Free allocated memory.
    hipFree(dev_c);

    return EXIT_SUCCESS;
}
